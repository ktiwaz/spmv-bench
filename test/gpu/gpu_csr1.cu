#include "hip/hip_runtime.h"
// A simple test executable for testing
// the COO loading functionality of the matrix
#include <iostream>
#include <string>
#include <cstdlib>

#include <spmm.h>
#include <csr.h>

__global__ void cuda_compute_csr(double *C, double *B, size_t rows, size_t cols, uint64_t *rowptr, uint64_t *rowidx, double *values) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < rows) {
        for (uint64_t p = rowptr[i]; p<rowptr[i+1]; p++) {
            uint64_t j = rowidx[p];
            for (uint64_t k = 0; k<rows; k++) {
                C[i*cols+k] += values[p] * B[j*cols+k];
            }
        }
    }
}

class CSR2 : public CSR {
public:
    explicit CSR2(std::string input) : CSR(input) {}

    double calculate() override {
        double start = getTime();
        
        double *dB, *dC, *dValues;
        uint64_t *dRowptr, *dRowidx;
        hipMalloc((void**)&dB, sizeof(double)*(rows*cols));
        hipMalloc((void**)&dC, sizeof(double)*(rows*cols));
        hipMalloc((void**)&dValues, sizeof(double)*(coo->nnz));
        hipMalloc((void**)&dRowptr, sizeof(uint64_t)*(rows+1));
        hipMalloc((void**)&dRowidx, sizeof(uint64_t)*(coo->nnz));
        
        hipMemcpy(dB, B, sizeof(double)*(rows*cols), hipMemcpyHostToDevice);
        hipMemcpy(dC, C, sizeof(double)*(rows*cols), hipMemcpyHostToDevice);
        hipMemcpy(dValues, values, sizeof(double)*(coo->nnz), hipMemcpyHostToDevice);
        hipMemcpy(dRowptr, rowptr, sizeof(uint64_t)*(rows+1), hipMemcpyHostToDevice);
        hipMemcpy(dRowidx, rowidx, sizeof(uint64_t)*(coo->nnz), hipMemcpyHostToDevice);
        
        cuda_compute_csr<<<16,1>>>(dC, dB, rows, cols, dRowptr, dRowidx, dValues);
        
        hipMemcpy(C, dC, sizeof(double)*(rows*cols), hipMemcpyDeviceToHost);
        
        hipFree(dB);
        hipFree(dC);
        hipFree(dValues);
        hipFree(dRowptr);
        hipFree(dRowidx);
        
        double end = getTime();
        return (double)(end-start);
    }
};

int main(int argc, char **argv) {
    CSR2 mtx(argv[1]);
    
    mtx.printSparse(false);
    std::cout << "-----------------" << std::endl;
    mtx.printDense(false);
    std::cout << "-----------------" << std::endl;
    double time = mtx.calculate();
    fprintf(stdout, "%lf\n", time);
    mtx.printResult(false);
    std::cout << "-----------------" << std::endl;
    
    return 0;
}

