/*
 * This kernel uses a global memory approach, where each thread is responsible for computing a single element in the resulting matrix. 
 * The use of thread and block indices to calculate the row and column indices helps in mapping the threads to the elements of the matrices.
 *
 */
 
#include <hip/hip_runtime.h>
#include <stdio.h>
 #include <stdlib.h>

#define BLOCK_SIZE 8

#define doubleW double

__global__ void global_element(double* A, double* B, double* C, int n) {

    double C_value = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    for (int k = 0; k < n; k++) {
        C_value += A[row * n + k] * B[n * k + col];
    }

    // Each thread writes one element to C matrix
    C[row * n + col] = C_value;
}

void matmul_kernel(size_t N, double* A, double* B, double* C) {
    double *A_device, *B_device, *C_device;
    hipMallocManaged(&A_device, N*N*sizeof(double));
    hipMallocManaged(&B_device, N*N*sizeof(double));
    hipMallocManaged(&C_device, N*N*sizeof(double));

    hipMemcpy(A_device, A, N*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(B_device, B, N*N*sizeof(double), hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(N / dimBlock.x, N / dimBlock.y);
    global_element<<<dimGrid, dimBlock>>>(A_device, B_device, C_device, N);
    hipDeviceSynchronize();

    hipMemcpy(C, C_device, N*N*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(A_device);
    hipFree(B_device);
    hipFree(C_device);
}

int main() {
    size_t N = 2048 * 4;
    double *A = (double *)malloc(sizeof(double)*N*N);
    double *B = (double *)malloc(sizeof(double)*N*N);
    double *C = (double *)malloc(sizeof(double)*N*N);
    for (size_t i = 0; i<N; i++) {
        A[i] = rand() % 101;
        B[i] = 1.7;
        C[i] = 0;
    }
    
    matmul_kernel(N, A, B, C);
}

